#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <cstring>

#define BLOCK_SIZE 16
#define DATA_SIZE_MB 100
#define NUM_BLOCKS (DATA_SIZE_MB * 1024 * 1024 / 16)
#define ROUND_KEYS_SIZE 240
#define AES_ROUNDS 14

typedef unsigned char uint8_t;
using namespace std;

const uint8_t sbox[256] = {
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76,
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0,
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15,
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75,
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84,
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf,
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8,
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2,
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73,
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb,
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79,
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08,
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a,
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e,
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf,
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16
};

const uint8_t inv_sbox[256] = {
    0x52,0x09,0x6a,0xd5,0x30,0x36,0xa5,0x38,0xbf,0x40,0xa3,0x9e,0x81,0xf3,0xd7,0xfb,
    0x7c,0xe3,0x39,0x82,0x9b,0x2f,0xff,0x87,0x34,0x8e,0x43,0x44,0xc4,0xde,0xe9,0xcb,
    0x54,0x7b,0x94,0x32,0xa6,0xc2,0x23,0x3d,0xee,0x4c,0x95,0x0b,0x42,0xfa,0xc3,0x4e,
    0x08,0x2e,0xa1,0x66,0x28,0xd9,0x24,0xb2,0x76,0x5b,0xa2,0x49,0x6d,0x8b,0xd1,0x25,
    0x72,0xf8,0xf6,0x64,0x86,0x68,0x98,0x16,0xd4,0xa4,0x5c,0xcc,0x5d,0x65,0xb6,0x92,
    0x6c,0x70,0x48,0x50,0xfd,0xed,0xb9,0xda,0x5e,0x15,0x46,0x57,0xa7,0x8d,0x9d,0x84,
    0x90,0xd8,0xab,0x00,0x8c,0xbc,0xd3,0x0a,0xf7,0xe4,0x58,0x05,0xb8,0xb3,0x45,0x06,
    0xd0,0x2c,0x1e,0x8f,0xca,0x3f,0x0f,0x02,0xc1,0xaf,0xbd,0x03,0x01,0x13,0x8a,0x6b,
    0x3a,0x91,0x11,0x41,0x4f,0x67,0xdc,0xea,0x97,0xf2,0xcf,0xce,0xf0,0xb4,0xe6,0x73,
    0x96,0xac,0x74,0x22,0xe7,0xad,0x35,0x85,0xe2,0xf9,0x37,0xe8,0x1c,0x75,0xdf,0x6e,
    0x47,0xf1,0x1a,0x71,0x1d,0x29,0xc5,0x89,0x6f,0xb7,0x62,0x0e,0xaa,0x18,0xbe,0x1b,
    0xfc,0x56,0x3e,0x4b,0xc6,0xd2,0x79,0x20,0x9a,0xdb,0xc0,0xfe,0x78,0xcd,0x5a,0xf4,
    0x1f,0xdd,0xa8,0x33,0x88,0x07,0xc7,0x31,0xb1,0x12,0x10,0x59,0x27,0x80,0xec,0x5f,
    0x60,0x51,0x7f,0xa9,0x19,0xb5,0x4a,0x0d,0x2d,0xe5,0x7a,0x9f,0x93,0xc9,0x9c,0xef,
    0xa0,0xe0,0x3b,0x4d,0xae,0x2a,0xf5,0xb0,0xc8,0xeb,0xbb,0x3c,0x83,0x53,0x99,0x61,
    0x17,0x2b,0x04,0x7e,0xba,0x77,0xd6,0x26,0xe1,0x69,0x14,0x63,0x55,0x21,0x0c,0x7d
};

__constant__ uint8_t d_sbox[256];
__constant__ uint8_t d_inv_sbox[256];

//////////////////////////////////////////////////////
// Helper Functions for AES Transformations
//////////////////////////////////////////////////////

// SubBytes transformation: substitute each byte in the state using the S-box
__device__ void SubBytes(uint8_t* state) {
    for (int i = 0; i < 16; i++) {
        state[i] = d_sbox[state[i]];
    }
}

// InvSubBytes transformation: substitute each byte in the state using the inverse S-box
__device__ void InvSubBytes(uint8_t* state) {
    for (int i = 0; i < 16; i++) {
        state[i] = d_inv_sbox[state[i]];
    }
}

// ShiftRows transformation: cyclically shift rows of the state to the left
__device__ void ShiftRows(uint8_t* state) {
    uint8_t tmp;
    // Row 1: shift left by 1
    tmp = state[1]; state[1] = state[5]; state[5] = state[9]; state[9] = state[13]; state[13] = tmp;
    // Row 2: shift left by 2
    tmp = state[2]; state[2] = state[10]; state[10] = tmp;
    tmp = state[6]; state[6] = state[14]; state[14] = tmp;
    // Row 3: shift left by 3
    tmp = state[3]; state[3] = state[15]; state[15] = state[11]; state[11] = state[7]; state[7] = tmp;
}

// InvShiftRows transformation: cyclically shift rows of the state to the right
__device__ void InvShiftRows(uint8_t* state) {
    uint8_t tmp;
    // Row 1: shift right by 1
    tmp = state[13]; state[13] = state[9]; state[9] = state[5]; state[5] = state[1]; state[1] = tmp;
    // Row 2: shift right by 2
    tmp = state[2]; state[2] = state[10]; state[10] = tmp;
    tmp = state[6]; state[6] = state[14]; state[14] = tmp;
    // Row 3: shift right by 3
    tmp = state[3]; state[3] = state[7]; state[7] = state[11]; state[11] = state[15]; state[15] = tmp;
}

// AddRoundKey transformation: XOR the state with the round key
__device__ void AddRoundKey(uint8_t* state, const uint8_t* roundKey) {
    for (int i = 0; i < 16; i++) {
        state[i] ^= roundKey[i];
    }
}

// xtime: multiply by 2 in GF(2^8)
__device__ uint8_t xtime(uint8_t x) {
    return (x << 1) ^ ((x & 0x80) ? 0x1B : 0x00);
}

// multiply: multiply two bytes in GF(2^8)
__device__ uint8_t multiply(uint8_t x, uint8_t y) {
    uint8_t result = 0;
    while (y) {
        if (y & 1) result ^= x;
        x = xtime(x);
        y >>= 1;
    }
    return result;
}

// MixColumns transformation: mix each column of the state matrix (encryption)
__device__ void MixColumns(uint8_t* state) {
    uint8_t temp[4];
    for (int i = 0; i < 4; i++) {
        temp[0] = multiply(0x02, state[i*4 + 0]) ^ multiply(0x03, state[i*4 + 1]) ^ state[i*4 + 2] ^ state[i*4 + 3];
        temp[1] = state[i*4 + 0] ^ multiply(0x02, state[i*4 + 1]) ^ multiply(0x03, state[i*4 + 2]) ^ state[i*4 + 3];
        temp[2] = state[i*4 + 0] ^ state[i*4 + 1] ^ multiply(0x02, state[i*4 + 2]) ^ multiply(0x03, state[i*4 + 3]);
        temp[3] = multiply(0x03, state[i*4 + 0]) ^ state[i*4 + 1] ^ state[i*4 + 2] ^ multiply(0x02, state[i*4 + 3]);
        for (int j = 0; j < 4; j++)
            state[i*4 + j] = temp[j];
    }
}

// InvMixColumns transformation: mix each column of the state matrix (decryption)
__device__ void InvMixColumns(uint8_t* state) {
    uint8_t temp[4];
    for (int i = 0; i < 4; i++) {
        temp[0] = multiply(0x0e, state[i*4 + 0]) ^ multiply(0x0b, state[i*4 + 1]) ^ multiply(0x0d, state[i*4 + 2]) ^ multiply(0x09, state[i*4 + 3]);
        temp[1] = multiply(0x09, state[i*4 + 0]) ^ multiply(0x0e, state[i*4 + 1]) ^ multiply(0x0b, state[i*4 + 2]) ^ multiply(0x0d, state[i*4 + 3]);
        temp[2] = multiply(0x0d, state[i*4 + 0]) ^ multiply(0x09, state[i*4 + 1]) ^ multiply(0x0e, state[i*4 + 2]) ^ multiply(0x0b, state[i*4 + 3]);
        temp[3] = multiply(0x0b, state[i*4 + 0]) ^ multiply(0x0d, state[i*4 + 1]) ^ multiply(0x09, state[i*4 + 2]) ^ multiply(0x0e, state[i*4 + 3]);
        for (int j = 0; j < 4; j++)
            state[i*4 + j] = temp[j];
    }
}

//////////////////////////////////////////////////////
// AES-256 Key Expansion (host-side)
//////////////////////////////////////////////////////

// KeyExpansion256: expands a 256-bit key into round keys for all AES rounds
void KeyExpansion256(const uint8_t* key, uint8_t* roundKeys) {
    const uint8_t Rcon[10] = {0x01,0x02,0x04,0x08,0x10,0x20,0x40,0x80,0x1B,0x36};
    memcpy(roundKeys, key, 32); // Copy original key as first 8 words (32 bytes)
    uint8_t temp[4];
    int i = 8;
    int rconIdx = 0;

    while (i < 60) { // AES-256 needs 60 words (4 bytes each)
        for (int j = 0; j < 4; j++)
            temp[j] = roundKeys[(i - 1) * 4 + j];

        if (i % 8 == 0) {
            // Rotate, substitute, and XOR with Rcon for every 8th word
            uint8_t t = temp[0];
            temp[0] = sbox[temp[1]] ^ Rcon[rconIdx++];
            temp[1] = sbox[temp[2]];
            temp[2] = sbox[temp[3]];
            temp[3] = sbox[t];
        } else if (i % 8 == 4) {
            // Substitute for every 4th word (except first)
            for (int j = 0; j < 4; j++)
                temp[j] = sbox[temp[j]];
        }

        // XOR with word 8 positions earlier
        for (int j = 0; j < 4; j++) {
            roundKeys[i * 4 + j] = roundKeys[(i - 8) * 4 + j] ^ temp[j];
        }
        i++;
    }
}

//////////////////////////////////////////////////////
// CUDA Kernels for AES-256 Encryption/Decryption
//////////////////////////////////////////////////////

// aes256_encrypt_kernel: encrypts each 16-byte block independently in parallel
__global__ void aes256_encrypt_kernel(uint8_t* input, uint8_t* output, const uint8_t* roundKeys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NUM_BLOCKS) return; // Out-of-bounds check

    uint8_t state[16];
    // Load input block into local state
    for (int i = 0; i < 16; i++) state[i] = input[idx * 16 + i];

    // Initial round key addition
    AddRoundKey(state, roundKeys);

    // Main AES rounds
    for (int round = 1; round < AES_ROUNDS; round++) {
        SubBytes(state);
        ShiftRows(state);
        MixColumns(state);
        AddRoundKey(state, roundKeys + round * 16);
    }
    // Final round (no MixColumns)
    SubBytes(state);
    ShiftRows(state);
    AddRoundKey(state, roundKeys + AES_ROUNDS * 16);

    // Store encrypted block to output
    for (int i = 0; i < 16; i++) output[idx * 16 + i] = state[i];
}

// aes256_decrypt_kernel: decrypts each 16-byte block independently in parallel
__global__ void aes256_decrypt_kernel(uint8_t* input, uint8_t* output, const uint8_t* roundKeys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NUM_BLOCKS) return; // Out-of-bounds check

    uint8_t state[16];
    // Load encrypted block into local state
    for (int i = 0; i < 16; i++) state[i] = input[idx * 16 + i];

    // Initial round key addition (last round key)
    AddRoundKey(state, roundKeys + AES_ROUNDS * 16);

    // Main AES rounds (in reverse)
    for (int round = AES_ROUNDS - 1; round > 0; round--) {
        InvShiftRows(state);
        InvSubBytes(state);
        AddRoundKey(state, roundKeys + round * 16);
        InvMixColumns(state);
    }
    // Final round (no InvMixColumns)
    InvShiftRows(state);
    InvSubBytes(state);
    AddRoundKey(state, roundKeys);

    // Store decrypted block to output
    for (int i = 0; i < 16; i++) output[idx * 16 + i] = state[i];
}

//////////////////////////////////////////////////////
// Main Program: AES-256 Batch Encryption/Decryption Test
//////////////////////////////////////////////////////
int main() {
    // Example 256-bit AES key (32 bytes)
    uint8_t key[32] = {
        0x60, 0x3d, 0xeb, 0x10, 0x15, 0xca, 0x71, 0xbe,
        0x2b, 0x73, 0xae, 0xf0, 0x85, 0x7d, 0x77, 0x81,
        0x1f, 0x35, 0x2c, 0x07, 0x3b, 0x61, 0x08, 0xd7,
        0x2d, 0x98, 0x10, 0xa3, 0x09, 0x14, 0xdf, 0xf4
    };

    // Allocate host memory for input, encrypted, and decrypted data
    uint8_t* h_input = new uint8_t[NUM_BLOCKS * BLOCK_SIZE];
    uint8_t* h_encrypted = new uint8_t[NUM_BLOCKS * BLOCK_SIZE];
    uint8_t* h_decrypted = new uint8_t[NUM_BLOCKS * BLOCK_SIZE];
    uint8_t roundKeys[ROUND_KEYS_SIZE];

    // Fill input with random data
    srand(12345);
    for (int i = 0; i < NUM_BLOCKS * BLOCK_SIZE; i++)
        h_input[i] = rand() % 256;

    // Expand the key for all AES rounds
    KeyExpansion256(key, roundKeys);

    // Copy S-boxes to GPU constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_sbox), sbox, 256);
    hipMemcpyToSymbol(HIP_SYMBOL(d_inv_sbox), inv_sbox, 256);

    // Allocate device memory
    uint8_t *d_input, *d_output, *d_decrypted, *d_roundKeys;
    hipMalloc(&d_input, NUM_BLOCKS * BLOCK_SIZE);
    hipMalloc(&d_output, NUM_BLOCKS * BLOCK_SIZE);
    hipMalloc(&d_decrypted, NUM_BLOCKS * BLOCK_SIZE);
    hipMalloc(&d_roundKeys, ROUND_KEYS_SIZE);

    // Set up CUDA kernel launch configuration
    dim3 blockDim(128); // 128 threads per block
    dim3 gridDim((NUM_BLOCKS + blockDim.x - 1) / blockDim.x); // Enough blocks to cover all data

    // CUDA events for timing
    hipEvent_t start, stop;
    float time_total = 0, time_H2D = 0, time_encrypt = 0, time_decrypt = 0, time_D2H = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    cout << "\n[GPU] AES-256 Batch Encryption: " << NUM_BLOCKS << " blocks\n";

    // Copy input data and round keys from host to device (timed)
    hipEventRecord(start);
    hipMemcpy(d_input, h_input, NUM_BLOCKS * BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_roundKeys, roundKeys, ROUND_KEYS_SIZE, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_H2D, start, stop);

    // Launch encryption kernel (timed)
    hipEventRecord(start);
    aes256_encrypt_kernel<<<gridDim, blockDim>>>(d_input, d_output, d_roundKeys);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_encrypt, start, stop);

    // Launch decryption kernel (timed)
    hipEventRecord(start);
    aes256_decrypt_kernel<<<gridDim, blockDim>>>(d_output, d_decrypted, d_roundKeys);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_decrypt, start, stop);

    // Copy results back from device to host (timed)
    hipEventRecord(start);
    hipMemcpy(h_encrypted, d_output, NUM_BLOCKS * BLOCK_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(h_decrypted, d_decrypted, NUM_BLOCKS * BLOCK_SIZE, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_D2H, start, stop);

    // Sum up total GPU time
    time_total = time_H2D + time_encrypt + time_decrypt + time_D2H;

    // Print timing breakdown
    printf("[GPU Timing Breakdown]\n");
    printf("  Host → Device Copy : %.3f ms\n", time_H2D);
    printf("  Encryption Kernel  : %.3f ms\n", time_encrypt);
    printf("  Decryption Kernel  : %.3f ms\n", time_decrypt);
    printf("  Device → Host Copy : %.3f ms\n", time_D2H);
    printf("  ------------------------------\n");
    printf("  Total GPU Time      : %.3f ms\n", time_total);

    // Verify that decrypted data matches original input
    bool match = true;
    for (int i = 0; i < NUM_BLOCKS * BLOCK_SIZE; i++) {
        if (h_input[i] != h_decrypted[i]) {
            printf("[✗] Mismatch at byte %d: input=%02x, decrypted=%02x\n", i, h_input[i], h_decrypted[i]);
            match = false;
            break;
        }
    }

    printf("%s\n", match ? "[✓] Decryption matches original plaintext." : "[✗] Decryption mismatch!");

    // Free device and host memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_decrypted);
    hipFree(d_roundKeys);
    delete[] h_input;
    delete[] h_encrypted;
    delete[] h_decrypted;

    return 0;
}