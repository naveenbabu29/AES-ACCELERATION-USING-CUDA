#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <cstring>
#include <chrono>
#include <thread>
#include <future>

#define BLOCK_SIZE 16
#define DATA_SIZE_MB 10
#define NUM_BLOCKS (DATA_SIZE_MB * 1024 * 1024 / 16)
#define ROUND_KEYS_SIZE 176

using namespace std;
typedef unsigned char uint8_t;

const uint8_t sbox[256] = {
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76,
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0,
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15,
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75,
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84,
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf,
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8,
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2,
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73,
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb,
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79,
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08,
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a,
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e,
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf,
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16
};

const uint8_t inv_sbox[256] = {
    0x52,0x09,0x6a,0xd5,0x30,0x36,0xa5,0x38,0xbf,0x40,0xa3,0x9e,0x81,0xf3,0xd7,0xfb,
    0x7c,0xe3,0x39,0x82,0x9b,0x2f,0xff,0x87,0x34,0x8e,0x43,0x44,0xc4,0xde,0xe9,0xcb,
    0x54,0x7b,0x94,0x32,0xa6,0xc2,0x23,0x3d,0xee,0x4c,0x95,0x0b,0x42,0xfa,0xc3,0x4e,
    0x08,0x2e,0xa1,0x66,0x28,0xd9,0x24,0xb2,0x76,0x5b,0xa2,0x49,0x6d,0x8b,0xd1,0x25,
    0x72,0xf8,0xf6,0x64,0x86,0x68,0x98,0x16,0xd4,0xa4,0x5c,0xcc,0x5d,0x65,0xb6,0x92,
    0x6c,0x70,0x48,0x50,0xfd,0xed,0xb9,0xda,0x5e,0x15,0x46,0x57,0xa7,0x8d,0x9d,0x84,
    0x90,0xd8,0xab,0x00,0x8c,0xbc,0xd3,0x0a,0xf7,0xe4,0x58,0x05,0xb8,0xb3,0x45,0x06,
    0xd0,0x2c,0x1e,0x8f,0xca,0x3f,0x0f,0x02,0xc1,0xaf,0xbd,0x03,0x01,0x13,0x8a,0x6b,
    0x3a,0x91,0x11,0x41,0x4f,0x67,0xdc,0xea,0x97,0xf2,0xcf,0xce,0xf0,0xb4,0xe6,0x73,
    0x96,0xac,0x74,0x22,0xe7,0xad,0x35,0x85,0xe2,0xf9,0x37,0xe8,0x1c,0x75,0xdf,0x6e,
    0x47,0xf1,0x1a,0x71,0x1d,0x29,0xc5,0x89,0x6f,0xb7,0x62,0x0e,0xaa,0x18,0xbe,0x1b,
    0xfc,0x56,0x3e,0x4b,0xc6,0xd2,0x79,0x20,0x9a,0xdb,0xc0,0xfe,0x78,0xcd,0x5a,0xf4,
    0x1f,0xdd,0xa8,0x33,0x88,0x07,0xc7,0x31,0xb1,0x12,0x10,0x59,0x27,0x80,0xec,0x5f,
    0x60,0x51,0x7f,0xa9,0x19,0xb5,0x4a,0x0d,0x2d,0xe5,0x7a,0x9f,0x93,0xc9,0x9c,0xef,
    0xa0,0xe0,0x3b,0x4d,0xae,0x2a,0xf5,0xb0,0xc8,0xeb,0xbb,0x3c,0x83,0x53,0x99,0x61,
    0x17,0x2b,0x04,0x7e,0xba,0x77,0xd6,0x26,0xe1,0x69,0x14,0x63,0x55,0x21,0x0c,0x7d
};

__constant__ uint8_t d_sbox[256];
__constant__ uint8_t d_inv_sbox[256];


// Helper functions

__device__ void SubBytes(uint8_t* state) {
    for (int i = 0; i < 16; i++) {
        state[i] = d_sbox[state[i]];
    }
}

__device__ void InvSubBytes(uint8_t* state) {
    for (int i = 0; i < 16; i++) {
        state[i] = d_inv_sbox[state[i]];
    }
}

__device__ void ShiftRows(uint8_t* state) {
    uint8_t tmp;
    tmp = state[1]; state[1] = state[5]; state[5] = state[9]; state[9] = state[13]; state[13] = tmp;
    tmp = state[2]; state[2] = state[10]; state[10] = tmp;
    tmp = state[6]; state[6] = state[14]; state[14] = tmp;
    tmp = state[3]; state[3] = state[15]; state[15] = state[11]; state[11] = state[7]; state[7] = tmp;
}

__device__ void InvShiftRows(uint8_t* state) {
    uint8_t tmp;
    tmp = state[13]; state[13] = state[9]; state[9] = state[5]; state[5] = state[1]; state[1] = tmp;
    tmp = state[2]; state[2] = state[10]; state[10] = tmp;
    tmp = state[6]; state[6] = state[14]; state[14] = tmp;
    tmp = state[3]; state[3] = state[7]; state[7] = state[11]; state[11] = state[15]; state[15] = tmp;
}

__device__ void AddRoundKey(uint8_t* state, uint8_t* roundKey) {
    for (int i = 0; i < 16; i++) {
        state[i] ^= roundKey[i];
    }
}

__device__ uint8_t xtime(uint8_t x) {
    return (x << 1) ^ ((x & 0x80) ? 0x1B : 0x00);
}

__device__ uint8_t multiply(uint8_t x, uint8_t y) {
    uint8_t result = 0;
    while (y) {
        if (y & 1) result ^= x;
        x = xtime(x);
        y >>= 1;
    }
    return result;
}

__device__ void MixColumns(uint8_t* state) {
    uint8_t temp[4];
    for (int i = 0; i < 4; i++) { // For each column
        temp[0] = multiply(0x02, state[i*4 + 0]) ^ multiply(0x03, state[i*4 + 1]) ^ state[i*4 + 2] ^ state[i*4 + 3];
        temp[1] = state[i*4 + 0] ^ multiply(0x02, state[i*4 + 1]) ^ multiply(0x03, state[i*4 + 2]) ^ state[i*4 + 3];
        temp[2] = state[i*4 + 0] ^ state[i*4 + 1] ^ multiply(0x02, state[i*4 + 2]) ^ multiply(0x03, state[i*4 + 3]);
        temp[3] = multiply(0x03, state[i*4 + 0]) ^ state[i*4 + 1] ^ state[i*4 + 2] ^ multiply(0x02, state[i*4 + 3]);
        for (int j = 0; j < 4; j++)
            state[i*4 + j] = temp[j];
    }
}

__device__ void InvMixColumns(uint8_t* state) {
    uint8_t temp[4];
    for (int i = 0; i < 4; i++) { // For each column
        temp[0] = multiply(0x0e, state[i*4 + 0]) ^ multiply(0x0b, state[i*4 + 1]) ^ multiply(0x0d, state[i*4 + 2]) ^ multiply(0x09, state[i*4 + 3]);
        temp[1] = multiply(0x09, state[i*4 + 0]) ^ multiply(0x0e, state[i*4 + 1]) ^ multiply(0x0b, state[i*4 + 2]) ^ multiply(0x0d, state[i*4 + 3]);
        temp[2] = multiply(0x0d, state[i*4 + 0]) ^ multiply(0x09, state[i*4 + 1]) ^ multiply(0x0e, state[i*4 + 2]) ^ multiply(0x0b, state[i*4 + 3]);
        temp[3] = multiply(0x0b, state[i*4 + 0]) ^ multiply(0x0d, state[i*4 + 1]) ^ multiply(0x09, state[i*4 + 2]) ^ multiply(0x0e, state[i*4 + 3]);
        for (int j = 0; j < 4; j++)
            state[i*4 + j] = temp[j];
    }
}


__global__ void aes128_encrypt_kernel(uint8_t* input, uint8_t* output, uint8_t* roundKeys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NUM_BLOCKS) return;

    uint8_t state[16];
    for (int i = 0; i < 16; i++) state[i] = input[idx * 16 + i];

    AddRoundKey(state, roundKeys);
    for (int round = 1; round < 10; round++) {
        SubBytes(state);
        ShiftRows(state);
        MixColumns(state);
        AddRoundKey(state, roundKeys + round * 16);
    }
    
    //final_round
    SubBytes(state);
    ShiftRows(state);
    AddRoundKey(state, roundKeys + 160);

    for (int i = 0; i < 16; i++) output[idx * 16 + i] = state[i];
}

__global__ void aes128_decrypt_kernel(uint8_t* input, uint8_t* output, uint8_t* roundKeys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NUM_BLOCKS) return;

    uint8_t state[16];
    for (int i = 0; i < 16; i++) state[i] = input[idx * 16 + i];

    AddRoundKey(state, roundKeys + 160);
    for (int round = 9; round > 0; round--) {
        InvShiftRows(state);
        InvSubBytes(state);
        AddRoundKey(state, roundKeys + round * 16);
        InvMixColumns(state);
    }
    //final round
    InvShiftRows(state);
    InvSubBytes(state);
    AddRoundKey(state, roundKeys);

    for (int i = 0; i < 16; i++) output[idx * 16 + i] = state[i];
}

void KeyExpansion(const uint8_t* key, uint8_t* roundKeys) {
    // Proper round constants for AES-128
    const uint8_t Rcon[10] = {0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1B, 0x36};
    
    // Copy the initial key (first 16 bytes)
    memcpy(roundKeys, key, 16);
    
    // Expand the key for the remaining 160 bytes (10 rounds, 16 bytes per round)
    for (int i = 4; i < 44; i++) {
        // Process one word (4 bytes) at a time
        uint8_t temp[4];
        
        // Get the previous word
        for (int j = 0; j < 4; j++) {
            temp[j] = roundKeys[(i-1)*4 + j];
        }
        
        // At the beginning of each round key (every 4th word)
        if (i % 4 == 0) {
            // RotWord - rotate left by one byte
            uint8_t t = temp[0];
            temp[0] = temp[1];
            temp[1] = temp[2];
            temp[2] = temp[3];
            temp[3] = t;
            
            // SubWord - substitute each byte using S-box
            for (int j = 0; j < 4; j++) {
                temp[j] = sbox[temp[j]];
            }
            
            // XOR with round constant (only first byte)
            temp[0] ^= Rcon[i/4 - 1];
        }
        
        // XOR with the word 4 positions back
        for (int j = 0; j < 4; j++) {
            roundKeys[i*4 + j] = roundKeys[(i-4)*4 + j] ^ temp[j];
        }
    }
}

void ascii_bomb(bool defused) {
    if (defused) {
        cout << "\n💣 Bomb Defused Just in Time!\n";
        cout << "    _____________\n";
        cout << "   |             |\n";
        cout << "   |   [SAFE]    |\n";
        cout << "   |_____________|\n";
    } else {
        cout << "\n💥 BOOM! The Bomb Exploded!\n";
        cout << "    _____________\n";
        cout << "   |             |\n";
        cout << "   |   [BOOM!]   |\n";
        cout << "   |_____________|\n";
    }
}


int main() {
    uint8_t* h_input = new uint8_t[NUM_BLOCKS * BLOCK_SIZE];
    uint8_t* h_output = new uint8_t[NUM_BLOCKS * BLOCK_SIZE];
    uint8_t* h_decrypted = new uint8_t[NUM_BLOCKS * BLOCK_SIZE];
    uint8_t key[16] = {
        0x2b, 0x7e, 0x15, 0x16,
        0x28, 0xae, 0xd2, 0xa6,
        0xab, 0xf7, 0x15, 0x88,
        0x09, 0xcf, 0x4f, 0x3c
    };
    uint8_t roundKeys[ROUND_KEYS_SIZE];
    KeyExpansion(key, roundKeys);
    hipMemcpyToSymbol(HIP_SYMBOL(d_sbox), sbox, 256);
    hipMemcpyToSymbol(HIP_SYMBOL(d_inv_sbox), inv_sbox, 256);

    srand(time(0));
    for (int i = 0; i < NUM_BLOCKS * BLOCK_SIZE; i++)
        h_input[i] = rand() % 256;

    uint8_t *d_input, *d_output, *d_decrypted, *d_roundKeys;
    hipMalloc(&d_input, NUM_BLOCKS * BLOCK_SIZE);
    hipMalloc(&d_output, NUM_BLOCKS * BLOCK_SIZE);
    hipMalloc(&d_decrypted, NUM_BLOCKS * BLOCK_SIZE);
    hipMalloc(&d_roundKeys, ROUND_KEYS_SIZE);

    hipMemcpy(d_input, h_input, NUM_BLOCKS * BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_roundKeys, roundKeys, ROUND_KEYS_SIZE, hipMemcpyHostToDevice);

    dim3 blockDim(256);
    dim3 gridDim((NUM_BLOCKS + blockDim.x - 1) / blockDim.x);

    const int countdown_ms = 500; // Bomb explodes after 150ms
    bool gpu_done = false;
    std::promise<float> time_result;
    std::future<float> time_future = time_result.get_future();

    auto bomb_thread = std::async(std::launch::async, [&]() {
        using namespace std::chrono_literals;
        for (int i = countdown_ms; i > 0; i -= 25) {
            if (gpu_done) return;
            cout << "[TIMER] " << i << " ms left...\n";
            this_thread::sleep_for(25ms);
        }
    });

    // Begin GPU encryption + decryption
    auto gpu_task = std::async(std::launch::async, [&]() {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        aes128_encrypt_kernel<<<gridDim, blockDim>>>(d_input, d_output, d_roundKeys);
        aes128_decrypt_kernel<<<gridDim, blockDim>>>(d_output, d_decrypted, d_roundKeys);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float total_time;
        hipEventElapsedTime(&total_time, start, stop);
        time_result.set_value(total_time);

        hipMemcpy(h_decrypted, d_decrypted, NUM_BLOCKS * BLOCK_SIZE, hipMemcpyDeviceToHost);
        gpu_done = true;

        bool match = true;
        for (int i = 0; i < NUM_BLOCKS * BLOCK_SIZE; ++i) {
            if (h_input[i] != h_decrypted[i]) {
                match = false;
                break;
            }
        }

        return match;
    });

    bool decrypted_correct = gpu_task.get();
    float elapsed_gpu_time = time_future.get();
    cout << "\n[GPU] Encryption + Decryption Time: " << elapsed_gpu_time << " ms\n";

    bool defused = decrypted_correct && (elapsed_gpu_time <= countdown_ms);
    ascii_bomb(defused);

    hipFree(d_input); hipFree(d_output); hipFree(d_decrypted); hipFree(d_roundKeys);
    delete[] h_input; delete[] h_output; delete[] h_decrypted;
    return 0;
}
